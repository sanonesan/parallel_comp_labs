#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include <cstddef>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <functional>
#include <iomanip>
#include <iostream>
#include <string>
#include <typeinfo>
#include <vector>

#include "config.h"


typedef float T;
typedef float4 T3;
const std::string T_TYPE_NAME = "float";
#define EPS 1e-6f

// typedef double T;
// typedef double4 T3;
// const std::string T_TYPE_NAME = "double";
// #define EPS 1e-6

#define BLOCK_SIZE 64
#define G T(6.67e-11)


void cudaCheckError(hipError_t error) {
	if (error != hipSuccess)
		printf("Error %d: %s\n", error, hipGetErrorString(error));
	// printf("Error %d: %s (file %s, line %d)\n", error,
	// hipGetErrorString(error), __FILE__, __LINE__);
}

void read_data(const std::string& path, std::vector<T>& m, std::vector<T3>& r,
			   std::vector<T3>& v) {
	std::ifstream fout(path);

	if (!fout.is_open()) {
		std::cout << "file is't open" << std::endl;
		throw 1;
	}

	std::size_t size = 0;
	fout >> size;

	m.resize(size);
	r.resize(size);
	v.resize(size);

	for (size_t i = 0; i < size; ++i) {
		fout >> m[i] >> r[i].x >> r[i].y >> r[i].z >> v[i].x >> v[i].y >>
			v[i].z;
	}

	fout.close();
}

/**
 * Functon for writing trajectories of bodies into file
 */
void write_bodies_coords(std::ostream& fout, const T time, const T3* r,
						 const int size) {
	fout << std::scientific;
	fout << std::setprecision(16);
	fout << time << ";";
	for (size_t i = 0; i < size; ++i) {
		fout << r[i].x << ";" << r[i].y << ";" << r[i].z << ";";
	}
	fout << std::endl;
}


/**
 * BEGIN BLOCK
 * Kernel functions for Mult and add
 */


__global__ void mult(T3* vec, T alpha, int size) {
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		vec[i].x *= alpha;
		vec[i].y *= alpha;
		vec[i].z *= alpha;
	}
}
__global__ void mult(T3* vec, T alpha, T3* res, int size) {
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		res[i].x = alpha * vec[i].x;
		res[i].y = alpha * vec[i].y;
		res[i].z = alpha * vec[i].z;
	}
}

__global__ void add(T3* vec1, T3* vec2, T3* res, int size) {
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		res[i].x = vec1[i].x + vec2[i].x;
		res[i].y = vec1[i].y + vec2[i].y;
		res[i].z = vec1[i].z + vec2[i].z;
	}
}

__global__ void add(T3* vec1, T3* vec2, int size) {
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		vec1[i].x += vec2[i].x;
		vec1[i].y += vec2[i].y;
		vec1[i].z += vec2[i].z;
	}
}

__global__ void mult_n_add(T3* vec1, T alpha, T3* vec2, T3* vec_res, int size) {
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		vec_res[i].x = vec1[i].x + alpha * vec2[i].x;
		vec_res[i].y = vec1[i].y + alpha * vec2[i].y;
		vec_res[i].z = vec1[i].z + alpha * vec2[i].z;
	}
}

/**
 * END BLOCK
 * Kernel functions for Mult and add
 */

__global__ void calc_acceleration(T* m, T3* r, T3* v, int size) {
	unsigned int globIdx = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int locIdx = threadIdx.x;

	T3 r_glob_tmp;
	if (globIdx < size) {
		r_glob_tmp = r[globIdx];
	} else {
		return;
	}

	__shared__ T m_shared[BLOCK_SIZE];
	__shared__ T3 r_shared[BLOCK_SIZE];


	T3 ri_rj;
	T den;
	T3 a_tmp{
		0.,
		0.,
		0.,
	};
	for (int i = 0; i < size; i += BLOCK_SIZE) {
		m_shared[locIdx] = m[i + locIdx];
		r_shared[locIdx] = r[i + locIdx];

		__syncthreads();

		for (int j = 0; j < BLOCK_SIZE; ++j) {
			if (i + j < size) {
				ri_rj.x = r_glob_tmp.x - r_shared[j].x;
				ri_rj.y = r_glob_tmp.y - r_shared[j].y;
				ri_rj.z = r_glob_tmp.z - r_shared[j].z;

				den = ri_rj.x * ri_rj.x + ri_rj.y * ri_rj.y + ri_rj.z * ri_rj.z;

				// if T == float

				den = sqrtf(den);
				den = den * den * den;
				den = __fdividef(m_shared[j], fmax(den, EPS * EPS * EPS));

				// if T == double

				// den = sqrt(den);
				// den = den * den * den;
				// den = m_shared[j] / fmax(den, EPS * EPS * EPS);

				a_tmp.x += ri_rj.x * den;
				a_tmp.y += ri_rj.y * den;
				a_tmp.z += ri_rj.z * den;
			}
		}

		__syncthreads();
	}
	if (globIdx < size) {
		v[globIdx].x = (-G) * a_tmp.x;
		v[globIdx].y = (-G) * a_tmp.y;
		v[globIdx].z = (-G) * a_tmp.z;
	}
}


// RHS function for Nbody problem
void Nbody_func_async(hipStream_t CUDA_STREAM, T* m, T3* r, T3* v, T3* r_res,
					  T3* v_res, dim3 threadsPerBlock, dim3 blocksInGrid,
					  int size) {
	hipMemcpyAsync(r_res, v, sizeof(T3) * size, hipMemcpyDeviceToDevice,
					CUDA_STREAM);
	calc_acceleration<<<blocksInGrid, threadsPerBlock, 0, CUDA_STREAM>>>(
		m, r, v_res, size);
}

void ode_RK2_CUDA(const std::vector<T>& m, const std::vector<T3>& r,
				  const std::vector<T3>& v, T t_start, T t_final, T tau,
				  std::string path = "", std::string path_1 = "") {
	std::size_t size = m.size();

	// output info about algorithm execution
	std::ofstream fout;
	// output for 4body test
	std::ofstream fout_1;


	/*
	 * Output for collecting info about calculations
	 */
	if (!path.empty()) fout = std::ofstream(path);
	if (!path.empty() && !fout.is_open()) {
		std::cout << "Could not open file for writing" << std::endl;
	} else {
		fout << "type;n_bodies;n_steps;block_size;time_taken;avg_exec_time\n";
	}

	/**
	 * Output for 4Body test
	 */
	if (!path_1.empty()) fout_1 = std::ofstream(path_1);
	if (!path_1.empty() && !fout_1.is_open()) {
		std::cout << "Could not open file for writing" << std::endl;
	} else {
		fout_1 << "time;";
		// r11;r12;r13;r21;r22;r23;r31;r32;r33;\n";
		for (int i = 1; i < size; ++i) {
			for (auto j : {1, 2, 3}) {
				fout_1 << "r_" + std::to_string(i) + "_" + std::to_string(j) +
							  ";";
			}
		}
		fout_1 << "r_" + std::to_string(size) + "_" + std::to_string(1) + ";";
		fout_1 << "r_" + std::to_string(size) + "_" + std::to_string(2) + ";";
		fout_1 << "r_" + std::to_string(size) + "_" + std::to_string(3) + "\n";
	}

	dim3 threadsPerBlock = BLOCK_SIZE;
	dim3 blocksInGrid = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

	// tmp vector of T3 type for 4body test output
	std::vector<T3> tmp;
	tmp.resize(size);

	// Creating CUDA stream for acync hipMemcpy (hipMemcpyAsync)
	hipStream_t CUDA_STREAM;
	hipStreamCreate(&CUDA_STREAM);

	// Masses, Radii and Velocities strored in the GPU memory
	T* m_cuda;
	T3 *r_cuda, *v_cuda;
	cudaCheckError(hipMalloc(&m_cuda, sizeof(T) * size));
	cudaCheckError(hipMalloc(&r_cuda, sizeof(T3) * size));
	cudaCheckError(hipMalloc(&v_cuda, sizeof(T3) * size));


	// Coefficient for RK2 algorithm
	T3 *k_r_cuda, *k_v_cuda;

	cudaCheckError(hipMalloc(&k_r_cuda, sizeof(T3) * size));
	cudaCheckError(hipMalloc(&k_v_cuda, sizeof(T3) * size));

	// Temporary Masses, Radii and Velocities strored in the GPU memory
	T3 *r_tmp_cuda, *v_tmp_cuda;
	cudaCheckError(hipMalloc(&r_tmp_cuda, sizeof(T3) * size));
	cudaCheckError(hipMalloc(&v_tmp_cuda, sizeof(T3) * size));


	// Copy from host to device
	cudaCheckError(
		hipMemcpy(m_cuda, m.data(), sizeof(T) * size, hipMemcpyHostToDevice));

	cudaCheckError(hipMemcpy(r_cuda, r.data(), sizeof(T3) * size,
							  hipMemcpyHostToDevice));

	cudaCheckError(hipMemcpy(v_cuda, v.data(), sizeof(T3) * size,
							  hipMemcpyHostToDevice));


	// Set time for the first step of RK2
	T t = t_start + tau;
	// Setting timer to measure time of calculations
	hipEvent_t timer_start, timer_stop;
	cudaCheckError(hipEventCreate(&timer_start));
	cudaCheckError(hipEventCreate(&timer_stop));
	// Start timer
	hipEventRecord(timer_start);
	while (true) {
		/**
		 * RK2 algorithm
		 */
		Nbody_func_async(CUDA_STREAM, m_cuda, r_cuda, v_cuda, k_r_cuda,
						 k_v_cuda, threadsPerBlock, blocksInGrid, size);
		mult_n_add<<<blocksInGrid, threadsPerBlock, 0, CUDA_STREAM>>>(
			r_cuda, tau / 2, k_r_cuda, r_tmp_cuda, size);
		mult_n_add<<<blocksInGrid, threadsPerBlock, 0, CUDA_STREAM>>>(
			v_cuda, tau / 2, k_v_cuda, v_tmp_cuda, size);

		Nbody_func_async(CUDA_STREAM, m_cuda, r_tmp_cuda, v_tmp_cuda, k_r_cuda,
						 k_v_cuda, threadsPerBlock, blocksInGrid, size);

		mult_n_add<<<blocksInGrid, threadsPerBlock, 0, CUDA_STREAM>>>(
			r_cuda, tau, k_r_cuda, r_cuda, size);
		mult_n_add<<<blocksInGrid, threadsPerBlock, 0, CUDA_STREAM>>>(
			v_cuda, tau, k_v_cuda, v_cuda, size);

		/**
		 * Commented code is another implementation of RK2 algorithm,
		 * where mult and add functions aren't gathered
		 */


		// Nbody_func_async(CUDA_STREAM, m_cuda, r_cuda, v_cuda, k_r_cuda,
		// 				 k_v_cuda, threadsPerBlock, blocksInGrid, size);
		//
		// mult<<<blocksInGrid, threadsPerBlock, 0, CUDA_STREAM>>>(k_r_cuda,
		// 														tau / 2, size);
		// mult<<<blocksInGrid, threadsPerBlock, 0, CUDA_STREAM>>>(k_v_cuda,
		// 														tau / 2, size);
		//
		// add<<<blocksInGrid, threadsPerBlock, 0, CUDA_STREAM>>>(
		// 	r_cuda, k_r_cuda, r_tmp_cuda, size);
		// add<<<blocksInGrid, threadsPerBlock, 0, CUDA_STREAM>>>(
		// 	v_cuda, k_v_cuda, v_tmp_cuda, size);
		//
		//
		// Nbody_func_async(CUDA_STREAM, m_cuda, r_tmp_cuda, v_tmp_cuda,
		// k_r_cuda, 				 k_v_cuda, threadsPerBlock, blocksInGrid,
		// size);
		//
		//
		// mult<<<blocksInGrid, threadsPerBlock, 0, CUDA_STREAM>>>(k_r_cuda,
		// tau, 														size);
		// mult<<<blocksInGrid, threadsPerBlock, 0, CUDA_STREAM>>>(k_v_cuda,
		// tau, 														size);
		// add<<<blocksInGrid, threadsPerBlock, 0, CUDA_STREAM>>>(r_cuda,
		// k_r_cuda, size); add<<<blocksInGrid, threadsPerBlock, 0,
		// CUDA_STREAM>>>(v_cuda,
		// k_v_cuda, size);

		if (!path_1.empty()) {
			/**
			 * Output for 4body test
			 */
			hipMemcpyAsync(tmp.data(), r_cuda, sizeof(T3) * size,
							hipMemcpyDeviceToHost, CUDA_STREAM);

			/**
			 * If output errors occured try this code instead of Acync.
			 * It should be slower, but it definitely works.
			 */
			// hipDeviceSynchronize();
			// hipMemcpy(tmp.data(), r_cuda, sizeof(T3) * size,
			// 				hipMemcpyDeviceToHost);
			write_bodies_coords(fout_1, t, tmp.data(), size);
		}

		if (t > t_final) break;
		t += tau;
	}
	// Stop timer
	hipEventRecord(timer_stop);
	hipEventSynchronize(timer_stop);

	// Calculate time taken for algorithm
	float elapsed_time = 0.;
	hipEventElapsedTime(&elapsed_time, timer_start, timer_stop);
	hipEventDestroy(timer_start);
	hipEventDestroy(timer_stop);
	// Calculate average time for on step of algorithm
	double avg_exec_time = elapsed_time / ((t_final - t_start) / tau);

	// Destroying variables stored in the GPU
	hipStreamDestroy(CUDA_STREAM);
	cudaCheckError(hipFree(m_cuda));
	cudaCheckError(hipFree(r_cuda));
	cudaCheckError(hipFree(v_cuda));

	cudaCheckError(hipFree(k_r_cuda));
	cudaCheckError(hipFree(k_v_cuda));

	cudaCheckError(hipFree(r_tmp_cuda));
	cudaCheckError(hipFree(v_tmp_cuda));

	// Output collected information of algorithm execution
	fout << T_TYPE_NAME + ";";
	fout << size << ";" << (int)((t_final - t_start) / tau) << ";" << BLOCK_SIZE
		 << ";" << elapsed_time / 1000. << ";" << avg_exec_time / 1000. << "\n";

	// Do not forget for close files
	fout.close();
	fout_1.close();

	return;
}

void test_4body() {
	std::vector<T> m;
	std::vector<T3> r, v;

	std::string read_path;
	std::string out_path;

	// testing on Personal Comuter

	// read_path = "./include/4body.txt";
	// read_data(read_path, m, r, v);
	//
	// out_path = "../output/res_4body_" + T_TYPE_NAME + "_" +
	// 		   std::to_string(BLOCK_SIZE) + "_.csv";
	//
	// ode_RK2_CUDA(m, r, v, 0., 20., 0.1, out_path,
	// 			 "../output/res4body_async.csv");

	// testing on claster

	read_path = config_lab_5_CUDA::PATH_src_folder + "/include/4body.txt";
	read_data(read_path, m, r, v);

	out_path = config_lab_5_CUDA::PATH_output_folder + "/res_4body_" +
			   T_TYPE_NAME + "_" + std::to_string(BLOCK_SIZE) + "_.csv";

	ode_RK2_CUDA(m, r, v, 0., 20., 0.1, out_path,
				 config_lab_5_CUDA::PATH_output_folder + "/res4body_async.csv");
}

void test_random_bodies() {
	std::vector<T> m;
	std::vector<T3> r, v;

	std::string read_path;
	std::string out_path;


	// testing on Personal Comuter

	// read_path = "./include/10k_bodies.txt";
	// read_data(read_path, m, r, v);
	//
	// out_path = "../output/res_10k_" + T_TYPE_NAME + "_" +
	// 		   std::to_string(BLOCK_SIZE) + "_.csv";

	// testing on claster

	read_path = config_lab_5_CUDA::PATH_src_folder + "/include/10k_bodies.txt";
	read_data(read_path, m, r, v);

	out_path = config_lab_5_CUDA::PATH_output_folder + "/res_10k_" +
			   T_TYPE_NAME + "_" + std::to_string(BLOCK_SIZE) + "_.csv";


	ode_RK2_CUDA(m, r, v, 0., 2., 0.1, out_path);


	read_path = config_lab_5_CUDA::PATH_src_folder + "/include/50k_bodies.txt";
	read_data(read_path, m, r, v);

	out_path = config_lab_5_CUDA::PATH_output_folder + "/res_50k_" +
			   T_TYPE_NAME + "_" + std::to_string(BLOCK_SIZE) + "_.csv";

	ode_RK2_CUDA(m, r, v, 0., 2., 0.1, out_path);


	read_path = config_lab_5_CUDA::PATH_src_folder + "/include/100k_bodies.txt";
	read_data(read_path, m, r, v);

	out_path = config_lab_5_CUDA::PATH_output_folder + "/res_100k_" +
			   T_TYPE_NAME + "_" + std::to_string(BLOCK_SIZE) + "_.csv";

	ode_RK2_CUDA(m, r, v, 0., 2., 0.1, out_path);


	read_path = config_lab_5_CUDA::PATH_src_folder + "/include/500k_bodies.txt";
	read_data(read_path, m, r, v);

	out_path = config_lab_5_CUDA::PATH_output_folder + "/res_500k_" +
			   T_TYPE_NAME + "_" + std::to_string(BLOCK_SIZE) + "_.csv";

	ode_RK2_CUDA(m, r, v, 0., 2., 0.1, out_path);
}

int main(int argc, char** argv) {
	test_4body();
	// test_random_bodies();
	return 0;
}
