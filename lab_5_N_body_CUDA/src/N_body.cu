#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
// Other
#include <stdio.h>

#include <cstddef>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <functional>
#include <iomanip>
#include <iostream>
#include <string>
#include <typeinfo>
#include <vector>

#include "config.h"

#define FLOAT__

#ifdef FLOAT__
typedef float T;
typedef float4 T3;

#else

typedef double T;
typedef double4 T3;

#endif

#define EPS T(1e-6)
#define BLOCK_SIZE 512
#define G T(6.67e-11)


void cudaCheckError(hipError_t error) {
	if (error != hipSuccess)
		printf("Error %d: %s\n", error, hipGetErrorString(error));
	// printf("Error %d: %s (file %s, line %d)\n", error,
	// hipGetErrorString(error), __FILE__, __LINE__);
}

T rand_num_type_T(T a, T b) {
	srand(time(NULL));
	return (b - a) * rand() / RAND_MAX + a;
}

void generate_data(std::size_t size, std::vector<T>& m, std::vector<T3>& r,
				   std::vector<T3>& v) {
	m.resize(size);
	r.resize(size);
	v.resize(size);

	for (std::size_t i = 0; i < size; ++i) {
		m[i] = rand_num_type_T(1e6, 1e8);

		r[i].x = rand_num_type_T(-1e2, 1e2);
		r[i].y = rand_num_type_T(-1e2, 1e2);
		r[i].z = rand_num_type_T(-1e2, 1e2);

		v[i].x = rand_num_type_T(-1e1, 1e1);
		v[i].y = rand_num_type_T(-1e1, 1e1);
		v[i].z = rand_num_type_T(-1e1, 1e1);
	}
}


void read_data(const std::string& path, std::vector<T>& m, std::vector<T3>& r,
			   std::vector<T3>& v) {
	std::ifstream fout(path);

	if (!fout.is_open()) {
		std::cout << "file is't open" << std::endl;
		throw 1;
	}

	std::size_t size = 0;
	fout >> size;

	m.resize(size);
	r.resize(size);
	v.resize(size);

	for (size_t i = 0; i < size; ++i) {
		fout >> m[i] >> r[i].x >> r[i].y >> r[i].z >> v[i].x >> v[i].y >>
			v[i].z;
	}

	fout.close();
}


void write_bodies_coords(std::ostream& fout, const T time, const T3* r,
						 const int size) {
	fout << std::scientific;
	fout << std::setprecision(16);
	fout << time << ";";
	for (size_t i = 0; i < size; ++i) {
		fout << r[i].x << ";" << r[i].y << ";" << r[i].z << ";";
	}
	fout << std::endl;
}

__global__ void mult(T3* vec, T alpha, int size) {
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		vec[i].x *= alpha;
		vec[i].y *= alpha;
		vec[i].z *= alpha;
	}
}
__global__ void mult(T3* vec, T alpha, T3* res, int size) {
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		res[i].x = alpha * vec[i].x;
		res[i].y = alpha * vec[i].y;
		res[i].z = alpha * vec[i].z;
	}
}

__global__ void add(T3* vec1, T3* vec2, T3* res, int size) {
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		res[i].x = vec1[i].x + vec2[i].x;
		res[i].y = vec1[i].y + vec2[i].y;
		res[i].z = vec1[i].z + vec2[i].z;
	}
}

__global__ void add(T3* vec1, T3* vec2, int size) {
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		vec1[i].x += vec2[i].x;
		vec1[i].y += vec2[i].y;
		vec1[i].z += vec2[i].z;
	}
}

__global__ void calc_acceleration(T* m, T3* r, T3* v, int size) {
	unsigned int globIdx = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int locIdx = threadIdx.x;

	__shared__ T m_shared[BLOCK_SIZE];
	__shared__ T3 r_shared[BLOCK_SIZE];

	T3 ri_rj;

	T3 th = r[globIdx];
	T den;

	T3 a_tmp{
		0.,
		0.,
		0.,
	};
	for (int i = 0; i < size; i += BLOCK_SIZE) {
		m_shared[locIdx] = m[i + locIdx];
		r_shared[locIdx] = r[i + locIdx];

		__syncthreads();

		for (int j = 0; j < BLOCK_SIZE; ++j) {
			if (i + j < size) {
				ri_rj.x = th.x - r_shared[j].x;
				ri_rj.y = th.y - r_shared[j].y;
				ri_rj.z = th.z - r_shared[j].z;

				den = ri_rj.x * ri_rj.x + ri_rj.y * ri_rj.y + ri_rj.z * ri_rj.z;
#ifdef FLOAT__
				den *= sqrtf(den);
				den = __fdividef(m_shared[j], fmax(den, EPS));
#else
				den *= sqrt(den);
				den = m_shared[j] / fmax(den, EPS);
#endif

				a_tmp.x += ri_rj.x * den;
				a_tmp.y += ri_rj.y * den;
				a_tmp.z += ri_rj.z * den;
			}
		}

		__syncthreads();
	}
	if (globIdx < size) {
		v[globIdx].x = (-G) * a_tmp.x;
		v[globIdx].y = (-G) * a_tmp.y;
		v[globIdx].z = (-G) * a_tmp.z;
	}
}

__global__ void mult_and_add(T3* vec1, T3* vec2, T3* res, T alpha, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		res[i].x = vec1[i].x + alpha * vec2[i].x;
		res[i].y = vec1[i].y + alpha * vec2[i].y;
		res[i].z = vec1[i].z + alpha * vec2[i].z;
	}
}

void Nbody_func(T* m, T3* r, T3* v, T3* r_res, T3* v_res, dim3 threadsPerBlock,
				dim3 blocksInGrid, int size) {
	hipMemcpy(r_res, v, sizeof(T3) * size, hipMemcpyDeviceToDevice);
	calc_acceleration<<<blocksInGrid, threadsPerBlock>>>(m, r, v_res, size);
}

void ode_RK2_CUDA(const std::vector<T>& m, const std::vector<T3>& r,
				  const std::vector<T3>& v, T t_start, T t_final, T tau,
				  std::string path = "") {
	std::size_t size = m.size();

	std::ofstream fout;
	//
	if (!path.empty()) fout = std::ofstream(path);
	//
	if (!path.empty() && !fout.is_open()) {
		std::cout << "Could not open file for writing" << std::endl;
	} else {
		fout << "type;n_bodies;n_steps;block_size:time_taken;avg_exec_time\n";
	}


	dim3 threadsPerBlock = BLOCK_SIZE;
	dim3 blocksInGrid = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;


	T* m_cuda;
	T3 *r_cuda, *v_cuda;
	cudaCheckError(hipMalloc(&m_cuda, sizeof(T) * size));
	cudaCheckError(hipMalloc(&r_cuda, sizeof(T3) * size));
	cudaCheckError(hipMalloc(&v_cuda, sizeof(T3) * size));


	T3 *k1_r_cuda, *k2_r_cuda, *k1_v_cuda, *k2_v_cuda;

	cudaCheckError(hipMalloc(&k1_r_cuda, sizeof(T3) * size));
	cudaCheckError(hipMalloc(&k2_r_cuda, sizeof(T3) * size));
	cudaCheckError(hipMalloc(&k1_v_cuda, sizeof(T3) * size));
	cudaCheckError(hipMalloc(&k2_v_cuda, sizeof(T3) * size));

	T3 *r_tmp_cuda, *v_tmp_cuda;
	cudaCheckError(hipMalloc(&r_tmp_cuda, sizeof(T3) * size));
	cudaCheckError(hipMalloc(&v_tmp_cuda, sizeof(T3) * size));


	// Copy from host to device
	cudaCheckError(
		hipMemcpy(m_cuda, m.data(), sizeof(T) * size, hipMemcpyHostToDevice));

	cudaCheckError(hipMemcpy(r_cuda, r.data(), sizeof(T3) * size,
							  hipMemcpyHostToDevice));

	cudaCheckError(hipMemcpy(v_cuda, v.data(), sizeof(T3) * size,
							  hipMemcpyHostToDevice));


	T t = t_start + tau;
	hipEvent_t timer_start, timer_stop;
	cudaCheckError(hipEventCreate(&timer_start));
	cudaCheckError(hipEventCreate(&timer_stop));

	hipEventRecord(timer_start);

	int n_steps = 0;
	while (true) {
		++n_steps;

		Nbody_func(m_cuda, r_cuda, v_cuda, k1_r_cuda, k1_v_cuda,
				   threadsPerBlock, blocksInGrid, size);

		mult<<<blocksInGrid, threadsPerBlock>>>(k1_r_cuda, tau / 2, size);
		mult<<<blocksInGrid, threadsPerBlock>>>(k1_v_cuda, tau / 2, size);

		add<<<blocksInGrid, threadsPerBlock>>>(r_cuda, k1_r_cuda, r_tmp_cuda,
											   size);
		add<<<blocksInGrid, threadsPerBlock>>>(v_cuda, k1_v_cuda, v_tmp_cuda,
											   size);


		Nbody_func(m_cuda, r_tmp_cuda, v_tmp_cuda, k1_r_cuda, k1_v_cuda,
				   threadsPerBlock, blocksInGrid, size);


		mult<<<blocksInGrid, threadsPerBlock>>>(k1_r_cuda, tau, size);
		mult<<<blocksInGrid, threadsPerBlock>>>(k1_v_cuda, tau, size);

		add<<<blocksInGrid, threadsPerBlock>>>(r_cuda, k1_r_cuda, size);
		add<<<blocksInGrid, threadsPerBlock>>>(v_cuda, k1_v_cuda, size);

		if (t > t_final) break;
		t += tau;
	}

	hipEventRecord(timer_stop);
	hipEventSynchronize(timer_stop);

	float elapsed_time = 0.;
	hipEventElapsedTime(&elapsed_time, timer_start, timer_stop);
	hipEventDestroy(timer_start);
	hipEventDestroy(timer_stop);
	double avg_exec_time = elapsed_time / ((t_final - t_start) / tau);


	cudaCheckError(hipFree(m_cuda));
	cudaCheckError(hipFree(r_cuda));
	cudaCheckError(hipFree(v_cuda));

	cudaCheckError(hipFree(k1_r_cuda));
	cudaCheckError(hipFree(k1_v_cuda));
	cudaCheckError(hipFree(k2_r_cuda));
	cudaCheckError(hipFree(k2_v_cuda));

	cudaCheckError(hipFree(r_tmp_cuda));
	cudaCheckError(hipFree(v_tmp_cuda));

#ifdef FLOAT__
	fout << "float;";
#else
	fout << "double;";
#endif

	fout << size << ";" << (int)((t_final - t_start) / tau) << ";" << BLOCK_SIZE
		 << ";" << elapsed_time / 1000. << ";" << avg_exec_time / 1000. << "\n";

	fout.close();

	return;
}

void tests() {
	std::vector<T> m;
	std::vector<T3> r, v;

	std::string read_path;
	std::string out_path;

	read_path = config_lab_5_CUDA::PATH_src_folder + "/include/10k_bodies.txt";
	read_data(read_path, m, r, v);

	out_path = config_lab_5_CUDA::PATH_output_folder + "/output/res_10k_" +
			   std::to_string(BLOCK_SIZE) + "_.csv";

	ode_RK2_CUDA(m, r, v, 0., 2., 0.1, out_path);

	read_path = config_lab_5_CUDA::PATH_src_folder + "/include/50k_bodies.txt";
	read_data(read_path, m, r, v);

	out_path = config_lab_5_CUDA::PATH_output_folder + "/output/res_50k_" +
			   std::to_string(BLOCK_SIZE) + "_.csv";

	ode_RK2_CUDA(m, r, v, 0., 2., 0.1, out_path);

	read_path = config_lab_5_CUDA::PATH_src_folder + "/include/100k_bodies.txt";
	read_data(read_path, m, r, v);

	out_path = config_lab_5_CUDA::PATH_output_folder + "/output/res_100k_" +
			   std::to_string(BLOCK_SIZE) + "_.csv";

	ode_RK2_CUDA(m, r, v, 0., 2., 0.1, out_path);

	read_path = config_lab_5_CUDA::PATH_src_folder + "/include/500k_bodies.txt";
	read_data(read_path, m, r, v);

	out_path = config_lab_5_CUDA::PATH_output_folder + "/output/res_500k_" +
			   std::to_string(BLOCK_SIZE) + "_.csv";

	ode_RK2_CUDA(m, r, v, 0., 2., 0.1, out_path);
}

int main(int argc, char** argv) {
	tests();
	return 0;
}
